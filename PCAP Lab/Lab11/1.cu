#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define MASK_WIDTH 5

__constant__ int M[MASK_WIDTH];
__global__ void oneDimConvKernel(int *d_arr, int *d_arr_output,  int arr_size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int start = index - (MASK_WIDTH - 1) / 2;
    int sum = 0;
    for (int j = 0; j < MASK_WIDTH; j++) {
        if (start + j >= 0 && start + j < arr_size) {
            sum += d_arr[start + j] * M[j];
        }
    }
    d_arr_output[index] = sum;
}

int main() {
    int mask[MASK_WIDTH] = {1, 2, 3, 2, 1};
    int arr_size = 7;
    int *arr = (int *) malloc(sizeof(int) * arr_size);
    int *arr_output = (int *) malloc(sizeof(int) * arr_size);

    printf("Enter %d elements of array:\n", arr_size);
    for (int i = 0; i < arr_size; i++) {
        scanf("%d", &arr[i]);
    }
    int threads_per_block = 256;
    int blocks_per_grid = (arr_size + threads_per_block - 1) / threads_per_block;
    dim3 dimBlock(threads_per_block, 1, 1);
    dim3 dimGrid(blocks_per_grid, 1, 1);
    int *d_arr, *d_arr_output;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    hipMalloc((void **) &d_arr, sizeof(int) * arr_size);
    hipMalloc((void **) &d_arr_output, sizeof(int) * arr_size);

    hipMemcpy(d_arr, arr, sizeof(int) * arr_size, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(M),mask, MASK_WIDTH*sizeof(int));
    oneDimConvKernel<<<dimGrid, dimBlock>>>( d_arr, d_arr_output, arr_size);

    hipMemcpy(arr_output, d_arr_output, sizeof(int) * arr_size, hipMemcpyDeviceToHost);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float time;
    hipEventElapsedTime(&time, start, stop);
    printf("Time taken for convolution: %f \n", time);
    printf("Array after Convolution:\n");
    for (int i = 0; i < arr_size; i++) {
        printf("%d ", arr_output[i]);
    }
    printf("\n");

    hipFree(d_arr);
    hipFree(d_arr_output);
    free(arr);
    free(arr_output);

    return 0;
}

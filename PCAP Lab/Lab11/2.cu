#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define MASK_WIDTH 5
#define TILE_SIZE 2

__constant__ int M[MASK_WIDTH];
__global__ void oneDimConvKernel(int *N, int *P, int Mask_Width, int Width)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float N_ds[TILE_SIZE + MASK_WIDTH - 1];
    int n = Mask_Width / 2;
    int halo_index_left = (blockIdx.x - 1) * blockDim.x + threadIdx.x;
    if (threadIdx.x >= blockDim.x - n)
    {
        N_ds[threadIdx.x - (blockDim.x - n)] =
            (halo_index_left < 0) ? 0 : N[halo_index_left];
    }
    N_ds[n + threadIdx.x] = N[blockIdx.x * blockDim.x + threadIdx.x];
    int halo_index_right = (blockIdx.x + 1) * blockDim.x + threadIdx.x;
    if (threadIdx.x < n)
    {
        N_ds[n + blockDim.x + threadIdx.x] =
            (halo_index_right >= Width) ? 0 : N[halo_index_right];
    }
    __syncthreads();
    int Pvalue = 0;
    for (int j = 0; j < Mask_Width; j++)
    {
        Pvalue += N_ds[threadIdx.x + j] * M[j];
    }
    P[i] = Pvalue;
}

int main()
{
    int mask[MASK_WIDTH] = {1, 2, 3, 2, 1};
    int arr_size = 12;
    int *arr = (int *)malloc(sizeof(int) * arr_size);
    int *arr_output = (int *)malloc(sizeof(int) * arr_size);

    printf("Enter %d elements of array:\n", arr_size);
    for (int i = 0; i < arr_size; i++)
    {
        scanf("%d", &arr[i]);
    }
    int threads_per_block = 256;
    int blocks_per_grid = (arr_size + threads_per_block - 1) / threads_per_block;
    dim3 dimBlock(threads_per_block, 1, 1);
    dim3 dimGrid(blocks_per_grid, 1, 1);
    int *d_arr, *d_arr_output;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    hipMalloc((void **)&d_arr, sizeof(int) * arr_size);
    hipMalloc((void **)&d_arr_output, sizeof(int) * arr_size);

    hipMemcpy(d_arr, arr, sizeof(int) * arr_size, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(M), mask, MASK_WIDTH * sizeof(int));
    oneDimConvKernel<<<4,3>>>(d_arr, d_arr_output, MASK_WIDTH, arr_size);

    hipMemcpy(arr_output, d_arr_output, sizeof(int) * arr_size, hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float time;
    hipEventElapsedTime(&time, start, stop);
    printf("Time taken for convolution: %f \n", time);
    printf("Array after Convolution:\n");
    for (int i = 0; i < arr_size; i++)
    {
        printf("%d ", arr_output[i]);
    }
    printf("\n");

    hipFree(d_arr);
    hipFree(d_arr_output);
    free(arr);
    free(arr_output);

    return 0;
}

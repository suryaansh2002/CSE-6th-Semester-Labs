#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__
void FindOccurrencesKernel(char *sentence, char *word, int length){
    int idx = threadIdx.x;
    int  size = length;
    int len = size - idx;
    int sumAll = size * (size+1)/2;
    int sumLess = len * (len+1)/2;
    int start = sumAll - sumLess;
    for (int i=0; i<len;i++){
        sentence[i+start]=word[i];
    }
}

int main(){
    printf("Enter length of word: ");
    int n;
    scanf("%d", &n);
    char *word = (char *)calloc(n, sizeof(char));
    printf("Enter the word: ");
    scanf("%s", word);
    char *final = (char *)calloc(n*(n+1)/2, sizeof(char));
    char *d_final, *d_word;
    hipMalloc((void **)&d_final, sizeof(char) * n * (n+1)/2);
    hipMalloc((void **)&d_word, sizeof(char) * n);
    hipMemcpy(d_final, final, sizeof(char) * (n+1) * n/2, hipMemcpyHostToDevice);
    hipMemcpy(d_word, word, sizeof(char) * n, hipMemcpyHostToDevice);
    FindOccurrencesKernel<<<1, n>>>(d_final, d_word, n);
    hipMemcpy(final, d_final, sizeof(char) * (n+1) * n/2, hipMemcpyDeviceToHost);
    printf("Output word is: %s\n", final);
    return 0;
}
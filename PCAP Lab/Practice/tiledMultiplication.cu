#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define WIDTH 4
#define TILE_WIDTH 2
__global__ tileMultiplication(int *M, int *N, int *P)
{
    __shared__ m_shared[TILE_WIDTH][TILE_WIDTH];
    __shared__ n_shared[TILE_WIDTH][TILE_WIDTH];

    int row = blockIdx.y * blockDim.y threadIdx.y;
    int col = blockIdx.x * blockDim.x threadIdx.x;

    int val = 0;
    int phases = WIDTH / TILE_WIDTH;
    for (int i = 0; i < phases; i++)
    {
        m_shared[threadIdx.y][threadIdx.x] = M[row * WIDTH + (i * TILE_WIDTH) + threadIdx.x];
        n_shared[threadIdx.y][threadIdx.x] = M[(threadIdx.y + (i * TILE_WIDTH)) * WIDTH + col];

        __syncthreads();
        for (int j = 0; j < TILE_WIDTH; j++)
        {
            val += m_shared[threadIdx.y][j] * n_shared[j][threadIdx.x];
        }
    }
    P[row * WIDTH + col] = val;
}
int main()
{
}
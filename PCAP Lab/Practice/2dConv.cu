#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void twoDimConvKernel(int *d_mask, int *d_arr, int *d_arr_output, int mask_width, int mask_height, int arr_width, int arr_height)
{
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    int startCol = col - (mask_width - 1) / 2;
    int startRow = row - (mask_height - 1) / 2;
    int sum = 0;
    for (int i = 0; i < mask_width; i++)
    {
        for (int j = 0; j < mask_height; j++)
        {

            if (startCol + i >= 0 && startCol + i < arr_width && startRow + j >= 0 && startRow + j < arr_height)
            {
                sum += d_arr[(startRow + j) * arr_width + (startCol + i)] * d_mask[j * mask_width + i];
            }
        }
    }

    d_arr_output[row * arr_width + col] = sum;
}

int main()
{
    int mask_width = 5;
    int mask_height = 7;
    int mask[mask_height][mask_width];

    // Populating mask matrix with elements from 1 to 35
    for (int i = 0; i < mask_height; i++)
    {
        for (int j = 0; j < mask_width; j++)
        {
            mask[i][j] = i * mask_width + j + 1;
        }
    }

    int arr_height = 10;
    int arr_width = 8;

    int *arr = (int *)malloc(sizeof(int) * arr_height * arr_width);
    int *arr_output = (int *)malloc(sizeof(int) * arr_height * arr_width);

    // Populating input array from 1 to 80
    for (int i = 0; i < arr_height * arr_width; i++)
    {
        arr[i] = i + 1;
    }

    dim3 dimBlock(arr_width, arr_height, 1);
    dim3 dimGrid(1, 1, 1);
    int *d_mask, *d_arr, *d_arr_output;
    hipMalloc((void **)&d_mask, sizeof(int) * mask_height * mask_width);
    hipMalloc((void **)&d_arr, sizeof(int) * arr_height * arr_width);
    hipMalloc((void **)&d_arr_output, sizeof(int) * arr_width * arr_height);

    hipMemcpy(d_mask, mask, sizeof(int) * mask_width * mask_height, hipMemcpyHostToDevice);
    hipMemcpy(d_arr, arr, sizeof(int) * arr_width * arr_height, hipMemcpyHostToDevice);

    twoDimConvKernel<<<dimGrid, dimBlock>>>(d_mask, d_arr, d_arr_output, mask_width, mask_height, arr_width, arr_height);
    hipMemcpy(arr_output, d_arr_output, sizeof(int) * arr_height * arr_width, hipMemcpyDeviceToHost);

    printf("Matrix after Convolution:\n");

    for (int i = 0; i < arr_height; i++)
    {
        for (int j = 0; j < arr_width; j++)
        {
            printf("%d ", arr_output[i * arr_width + j]);
        }
        printf("\n");
    }

    free(arr);
    free(arr_output);
    hipFree(d_mask);
    hipFree(d_arr);
    hipFree(d_arr_output);

    return 0;
}

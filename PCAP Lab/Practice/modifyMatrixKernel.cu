#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__host__ __device__ int isPrime(int x)
{
    for (int i = 2; i * i <= x; i++)
    {
        if (x % i == 0)
        {
            return 0;
        }
    }
    return 1;
}

__global__ void modifyMatrixKernel(int *d_matA, char *d_matB, int *d_n, int *d_m)
{
    int blockId = (blockIdx.z * gridDim.y * gridDim.x) + (blockIdx.y * gridDim.x) + blockIdx.x;
    int threadId = (blockId * blockDim.z * blockDim.y * blockDim.x) + (threadIdx.z * blockDim.y * blockDim.x) + (threadIdx.y * blockDim.x) + threadIdx.x;
    
    if (threadId < *d_n) // Since each thread does for one row
    {
        for (int k = 0; k < *d_m; k++)
        {
            char c;
            int ele = d_matA[threadId * (*d_m) + k];
            if (ele == 0)
            {
                c = 'Z';
            }
            else if (ele == 1)
            {
                c = 'O';
            }
            else if (ele < 0)
            {
                c = 'N';
            }
            else
            {
                if (isPrime(ele) == 1)
                {
                    c = 'P';
                }
                else
                {
                    c = 'C';
                }
            }
            d_matB[threadId * (*d_m) + k] = c;
        }
    }
}

int main()
{
    int n, m;
    printf("Enter number of rows and columns in matrix:");
    scanf("%d", &n);
    scanf("%d", &m);
    printf("Enter %d elements of the matrix:", n * m);
    int *mat = (int *)malloc(sizeof(int) * m * n);
    char *matB = (char *)malloc(sizeof(char) * m * n);

    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < m; j++)
        {
            scanf("%d", &mat[i * m + j]);
        }
    }
    int *d_mat, *d_n, *d_m;
    char *d_matB;
    hipMalloc((void **)&d_mat, sizeof(int) * n * m);
    hipMalloc((void **)&d_matB, sizeof(char) * n * m);
    hipMalloc((void **)&d_n, sizeof(int));
    hipMalloc((void **)&d_m, sizeof(int));

    hipMemcpy(d_mat, mat, sizeof(int) * n * m, hipMemcpyHostToDevice);
    hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_m, &m, sizeof(int), hipMemcpyHostToDevice);
    dim3 dimGrid(3, 4, 3);
    dim3 dimBlock(4, 3, 3);

    modifyMatrixKernel<<<dimGrid, dimBlock>>>(d_mat, d_matB, d_n, d_m);

    hipMemcpy(matB, d_matB, sizeof(char) * n * m, hipMemcpyDeviceToHost);

    printf("Output Matrix:\n");
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < m; j++)
        {
            printf("%c", matB[i * m + j]);
        }
    }
}
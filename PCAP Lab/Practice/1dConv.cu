#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void oneDimConvKernel(int *d_mask, int *d_arr, int *d_arr_output, int mask_width, int arr_size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int start = index - (mask_width - 1) / 2;
    int sum = 0;
    for (int j = 0; j < mask_width; j++) {
        if (start + j >= 0 && start + j < arr_size) {
            sum += d_arr[start + j] * d_mask[j];
        }
    }
    d_arr_output[index] = sum;
}

int main() {
    int mask[5] = {1, 2, 3, 2, 1};
    int mask_width = 5;
    int arr_size = 7;
    int *arr = (int *) malloc(sizeof(int) * arr_size);
    int *arr_output = (int *) malloc(sizeof(int) * arr_size);

    printf("Enter %d elements of array:\n", arr_size);
    for (int i = 0; i < arr_size; i++) {
        scanf("%d", &arr[i]);
    }
    int threads_per_block = 256;
    int blocks_per_grid = (arr_size + threads_per_block - 1) / threads_per_block;
    dim3 dimBlock(threads_per_block, 1, 1);
    dim3 dimGrid(blocks_per_grid, 1, 1);
    int *d_mask, *d_arr, *d_arr_output;

    hipMalloc((void **) &d_mask, sizeof(int) * mask_width);
    hipMalloc((void **) &d_arr, sizeof(int) * arr_size);
    hipMalloc((void **) &d_arr_output, sizeof(int) * arr_size);

    hipMemcpy(d_mask, mask, sizeof(int) * mask_width, hipMemcpyHostToDevice);
    hipMemcpy(d_arr, arr, sizeof(int) * arr_size, hipMemcpyHostToDevice);

    oneDimConvKernel<<<dimGrid, dimBlock>>>(d_mask, d_arr, d_arr_output, mask_width, arr_size);

    hipMemcpy(arr_output, d_arr_output, sizeof(int) * arr_size, hipMemcpyDeviceToHost);

    printf("Array after Convolution:\n");
    for (int i = 0; i < arr_size; i++) {
        printf("%d ", arr_output[i]);
    }
    printf("\n");

    hipFree(d_mask);
    hipFree(d_arr);
    hipFree(d_arr_output);

    free(arr);
    free(arr_output);

    return 0;
}

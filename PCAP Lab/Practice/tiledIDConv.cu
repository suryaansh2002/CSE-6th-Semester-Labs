#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define WIDTH 16
#define MASK_WIDTH 5
#define TILE_WIDTH 4
__constant__ M[MASK_WIDTH];
__global__ oneDTiledConv(int *N, int *P)
{
    __shared__ n_shared[TILE_WIDTH + MASK_WIDTH - 1];
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int n = MASK_WIDTH / 2;
    int halo_left = (blockIdx.x - 1) * blockDim.x + threadIdx.x;
    int halo_right = (blockIdx.x + 1) * blockDim.x + threadIdx.x;

    if (threadIdx.x > blockDim.x - n)
    {
        n_shared[threadIdx.x - (blockDim.x - n)] = halo_left < 0 ? 0 : N[halo_left];
    }
    n_shared[n + threadIdx.x] = N[id];
    if (threadIdx.x < n)
    {
        n_shared[threadIdx.x + blockDim.x + n] = halo_right > WIDTH ? 0 : N[halo_right];
    }
    __syncthreads();
    int val = 0;
    for (int j = 0; j < MASK_WIDTH; j++)
    {
        val += M[j] * n_shared[threadIdx.x + j];
    }
    P[i] = val;
}

int main()
{
}
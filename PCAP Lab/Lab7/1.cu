#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

// 1. Write and execute a program in CUDA to add two vectors of length N to meet the following requirements using 3 different kernels 

// a) block size as N 

// b) N threads within a block

// c) Keep the number of threads per block as 256 (constant) and vary the number of blocks to handle N elements.

__global__
void vecAddKernel(float *a,float *b,float *c){
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	c[idx] = a[idx]+b[idx];
}


void vecAdd(float *a,float *b,float *c,int n){
	int size = n * sizeof(float);
	float *d_a,*d_b,*d_c;
	hipMalloc((void**) &d_a,size);
	hipMalloc((void**) &d_b,size);
	hipMalloc((void**) &d_c,size);
	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
	printf("a:");
	for(int i = 0;i<n;i++){
		printf("%.2f,",a[i]);
	}
	printf("\n");
	printf("b:");
	for(int i = 0;i<n;i++){
		printf("%.2f,",b[i]);
	}
	printf("\n\n\n");
	vecAddKernel<<<n,1>>> (d_a,d_b,d_c);
	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	printf("a + b(block size n):");
	for(int i = 0;i<n;i++){
		printf("%.2f,",c[i]);
	}
	printf("\n\n");

	vecAddKernel<<<1,n>>> (d_a,d_b,d_c);
	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	printf("a + b(n threads):");
	for(int i = 0;i<n;i++){
		printf("%.2f,",c[i]);
	}
	printf("\n\n");	

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}

int main(){
	float *ha,*hb,*hc;
	int n = 5;
	int size = n*sizeof(float);
	ha = (float*)malloc(size);
	hb = (float*)malloc(size);
	hc = (float*)malloc(size);

	for(int i = 0;i<n;i++){
		ha[i] = (i+1)*2;
		hb[i] = (i+1);
	}
	vecAdd(ha,hb,hc,n);
	return 0;
}
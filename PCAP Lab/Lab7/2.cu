#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

// 1. Write and execute a program in CUDA to add two vectors of length N to meet the following requirements using 3 different kernels 

// a) block size as N 

// b) N threads within a block

// c) Keep the number of threads per block as 256 (constant) and vary the number of blocks to handle N elements.

__global__
void vecAddKernel(float *a,float *b,float *c,int n){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < n){
		c[idx] = a[idx] + b[idx];
	}
}

void vecAdd(float *a,float *b,float *c,int n){
	int size = n * sizeof(float);
	float *d_a,*d_b,*d_c;
	hipMalloc((void**) &d_a,size);
	hipMalloc((void**) &d_b,size);
	hipMalloc((void**) &d_c,size);

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
	printf("a:");
	for(int i = 0;i<n;i++){
		printf("%.2f,",a[i]);
	}
	printf("\n");
	printf("\n");
	printf("b:");
	for(int i = 0;i<n;i++){
		printf("%.2f,",b[i]);
	}
	printf("\n");
	printf("\n");
	vecAddKernel<<<((n+255)/256),256>>> (d_a,d_b,d_c,n);
	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	printf("a + b:");
	for(int i = 0;i<n;i++){
		printf("%.2f,",c[i]);
	}

	printf("\n");
	printf("\n");
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}

int main(){
	float *ha,*hb,*hc;
	int n = 5;
	int size = n*sizeof(float);
	ha = (float*)malloc(size);
	hb = (float*)malloc(size);
	hc = (float*)malloc(size);

	for(int i = 0;i<n;i++){
		ha[i] = (i+1)*2;
		hb[i] = (i+1);
	}
	vecAdd(ha,hb,hc,n);
	return 0;
}
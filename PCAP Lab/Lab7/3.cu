#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__
void vecAddKernel(float *a, float*b, float *c,int n, int m){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
    int sum=0;
    int mid= (m-1)/2;
    for(int j=0; j<m; j++){
        if((i-mid+j)>=0 && (i-mid+j)<n){
            sum= sum + (a[i-mid+j])*(b[j]);
        }
    }
	c[i] = sum;
}

void vecAdd(float *a,float *c, float *b,int n, int m){
	int size = n * sizeof(float);
	float *d_a,*d_b,*d_c;
	hipMalloc((void**) &d_a,size);
	hipMalloc((void**) &d_b,size);
	hipMalloc((void**) &d_c,size);

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,m * sizeof(float),hipMemcpyHostToDevice);

	printf("a:");
	for(int i = 0;i<n;i++){
		printf("%.2f,",a[i]);
	}
	printf("\n");
	printf("\n");
	vecAddKernel<<<n,1>>> (d_a,d_b, d_c,n, m);
	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	printf("c:");
	for(int i = 0;i<n;i++){
		printf("%.2f,",c[i]);
	}

	printf("\n");
	printf("\n");
	hipFree(d_a);
	hipFree(d_c);
}

int main(){
    int n=8,m=5;
	float ha[8]={1,2,3,4,5,6,7,8};
    float hb[8];
    float mask[5]={1,2,3,4,5};
	vecAdd(ha,hb,mask,n, m);
	return 0;
}
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

// 1. Write and execute a program in CUDA to add two vectors of length N to meet the following requirements using 3 different kernels 

// a) block size as N 

// b) N threads within a block

// c) Keep the number of threads per block as 256 (constant) and vary the number of blocks to handle N elements.

__global__
void vecAddKernel(float *a, float *c,int n){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	c[idx] = sin(a[idx]);
}

void vecAdd(float *a,float *c,int n){
	int size = n * sizeof(float);
	float *d_a,*d_c;
	hipMalloc((void**) &d_a,size);
	hipMalloc((void**) &d_c,size);

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	printf("a:");
	for(int i = 0;i<n;i++){
		printf("%.2f,",a[i]);
	}
	printf("\n");
	printf("\n");
	vecAddKernel<<<n,1>>> (d_a,d_c,n);
	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	printf("c:");
	for(int i = 0;i<n;i++){
		printf("%.2f,",c[i]);
	}

	printf("\n");
	printf("\n");
	hipFree(d_a);
	hipFree(d_c);
}

int main(){
	float *ha,*hc;
	int n = 5;
	int size = n*sizeof(float);
	ha = (float*)malloc(size);
	hc = (float*)malloc(size);
	for(int i = 0;i<n;i++){
		ha[i] = (i+1)*2;
	}
    ha[0]=(double)22/7;
	vecAdd(ha,hc,n);
	return 0;
}
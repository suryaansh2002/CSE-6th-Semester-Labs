#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__host__ __device__ void printMatrix(const char *string, int *A, int width)
{
    printf("%s\n", string);
    for (int i = 0; i < width; i++)
    {
        for (int j = 0; j < width; j++)
            printf("%d\t", A[i * width + j]);
        printf("\n");
    }
    printf("\n");
}
__host__ void clearMatrix(int *A, int width)
{
    for (int i = 0; i < width; i++)
    {
        for (int j = 0; j < width; j++)
            A[i * width + j] = 0;
    }
}
__global__ void addMatrixKernel_1a(int *A, int *B, int *C, int width)
{
    int row = threadIdx.y;
    for (int i = 0; i < width; i++)
    {
        int ind = row * width + i;
        C[ind] = A[ind] + B[ind];
    }
}
__global__ void addMatrixKernel_1b(int *A, int *B, int *C, int width)
{
    int col = threadIdx.x;
    for (int i = 0; i < width; i++)
    {
        int ind = i * width + col;
        C[ind] = A[ind] + B[ind];
    }
}
__global__ void addMatrixKernel_1c(int *A, int *B, int *C, int width)
{
    int col = threadIdx.x;
    int row = threadIdx.y;
    int ind=row * width + col;
    C[ind] = A[ind] + B[ind];
}
void addMatrix(int *h_A, int *h_B, int *h_C, int width)
{
    int *d_A, *d_B, *d_C;
    int size = width * width * sizeof(int);
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);
    dim3 dimBlock(1, 1, 1);
    dim3 dimGrid(1, 1, 1);
    dimBlock.x = 1;
    dimBlock.y = width;
    dimBlock.z = 1;
    addMatrixKernel_1a<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, width);
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    printMatrix("A+B: (from 1a kernel): ", h_C, width);
    clearMatrix(h_C, width);
    hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);
    dimBlock.x = width;
    dimBlock.y = 1;
    dimBlock.z = 1;
    addMatrixKernel_1b<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, width);
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    printMatrix("A+B: (from 1b kernel): ", h_C, width);
    clearMatrix(h_C, width);
    hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);
    dimBlock.x = width;
    dimBlock.y = width;
    dimBlock.z = 1;
    addMatrixKernel_1c<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, width);
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    printMatrix("A+B: (from 1c kernel): ", h_C, width);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
int main()
{
    int *A, *B, *C;
    int width;
    printf("Enter width: ");
    scanf("%d", &width);
    int size = width * width * sizeof(int);
    A = (int *)malloc(size);
    B = (int *)malloc(size);
    C = (int *)malloc(size);
    printf("Enter %d elements for A: ", (width * width));
    for (int i = 0; i < width; i++)
    {
        for (int j = 0; j < width; j++)
        {
            scanf("%d", &A[i * width + j]);
        }
    }
    printf("Enter %d elements for B: ", (width * width));
    for (int i = 0; i < width; i++)
    {
        for (int j = 0; j < width; j++)
        {
   
// Write a program in CUDA to multiply two Matrices for the following specifications:  
	 
// 	a. Each row of resultant matrix to be computed by one thread.  
	 
// 	b. Each column of resultant matrix to be computed by one thread.  
	 
// 	c. Each element of resultant matrix to be computed by one thread. 
         scanf("%d", &B[i * width + j]);
        }
    }
    printMatrix("A:", A, width);
    printMatrix("B:", B, width);
    addMatrix(A, B, C, width);
    return 0;
}
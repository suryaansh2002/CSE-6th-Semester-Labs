 #include<stdio.h>
 #include<hip/hip_runtime.h>
 #include<stdlib.h>
#include<math.h>

__global__ void modfiyMatrixKernel(int*d_mat, int* d_mat_output, int m, int n){
    int id=blockIdx.x*blockDim.x + threadIdx.x;
    if(id<m){
        for(int k=0; k<n; k++){
            d_mat_output[id*n + k]=pow(d_mat[id*n + k], id+1);
        }
    }

}
 int main(){

    printf("Enter the no of rows and cols of the matrix: ");
    int m,n;
    scanf("%d %d", &m, &n);
    printf("Enter %d elements of the matrix:\n", m*n);
    int* mat=(int *)malloc(sizeof(int)*n*m);
    int* mat_output=(int *)malloc(sizeof(int)*n*m);
    for(int i=0; i<m; i++){
        for(int j=0; j<n; j++){
            scanf("%d", &mat[i*n + j]);
        }
    }
    int*d_mat, *d_mat_output;

    hipMalloc((void **) &d_mat, sizeof(int)*n*m);
    hipMalloc((void **) &d_mat_output, sizeof(int)*n*m);

    hipMemcpy(d_mat, mat, sizeof(int)*n*m, hipMemcpyHostToDevice);
    modfiyMatrixKernel<<<1,m>>>(d_mat,d_mat_output, m,n);
    hipMemcpy(mat_output, d_mat_output, sizeof(int)*n*m, hipMemcpyDeviceToHost);
    printf("Output matrix is: \n");
    for(int i=0; i<m; i++){
        for(int j=0; j<n; j++){
            printf("%d ", mat_output[i*n + j]);
        }
        printf("\n");
    }
 }
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

__host__ __device__ int decToOnesComp(int n)
{
    int binaryNum[32];
    int i = 0;
    while (n > 0)
    {
        binaryNum[i] = n % 2;
        n = n / 2;
        i++;
    }
    int x = 0;
    for (int j = i - 1; j >= 0; j--)
    {
        x = x * 10;
        x += !binaryNum[j];
    }
    return x;
}
__global__ void modfiyMatrixKernel(int *d_mat, int m, int n)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x + 1;
    if (id < m)
    {
        
        for (int k = 1; k < n - 1; k++)
        {
            d_mat[id * n + k] = decToOnesComp(d_mat[id * n + k]);
        }
    }
}
int main()
{

    printf("Enter the no of rows and cols of the matrix: ");
    int m, n;
    scanf("%d %d", &m, &n);
    printf("Enter %d elements of the matrix:\n", m * n);
    int *mat = (int *)malloc(sizeof(int) * n * m);
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            scanf("%d", &mat[i * n + j]);
        }
    }
    int *d_mat;

    hipMalloc((void **)&d_mat, sizeof(int) * n * m);

    hipMemcpy(d_mat, mat, sizeof(int) * n * m, hipMemcpyHostToDevice);
    modfiyMatrixKernel<<<1, m-2 >>>(d_mat, m, n);
    hipMemcpy(mat, d_mat, sizeof(int) * n * m, hipMemcpyDeviceToHost);
    printf("Output matrix is: \n");
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            printf("%d ", mat[i * n + j]);
        }
        printf("\n");
    }
}